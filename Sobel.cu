#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include "mypgm.h"

#define SIZE 4096
#define N 16384
#define L 1024
#define INF (1<<30)
#define MIN_(a,b) (((a)<(b))?(a):(b))
#define MAX_(a,b) (((a)>(b))?(a):(b))
#define sqr(x) (x)*(x)

typedef float uc;
float minimo =  INF;
float maximo = -INF;

__global__ void kernel(uc *img1, uc *img2,float mini,float maxi){
	int offset = (blockIdx.x * L) + threadIdx.x;

	int left = offset - 1;
	int right = offset + 1;

	int top = offset - N;
	int bottom = offset + N;

	// Esquinas
	int sqrA = top-1;
	int sqrB = top+1;
	int sqrC = bottom-1;
	int sqrD = bottom+1;

	// Mascara Horizontal
	float p = 0.0;  
	p += -1 * img1[sqrA];
	p += 0  * img1[left];
	p += 1  * img1[sqrC];
	p += -2 * img1[top];
	p += 0  * img1[offset];
	p += 2  * img1[bottom];
	p += -1 * img1[sqrB];
	p += 0  * img1[right];
	p += 1  * img1[sqrD];
	p = (float)MAX_BRIGHTNESS * (float)(p - mini) / (float)(maxi - mini);

	// Mascara Vertical
	float q = 0.0;  
	q += -1 * img1[sqrA];
	q += -2 * img1[left];
	q += -1 * img1[sqrC];
	q += 0  * img1[top];
	q += 0  * img1[offset];
	q += 0  * img1[bottom];
	q += 1  * img1[sqrB];
	q += 2  * img1[right];
	q += 1  * img1[sqrD];

	q = (float)MAX_BRIGHTNESS * (float)(p - mini) / (float)(maxi - mini);

	img2[offset] = sqrt(sqr(q) + sqr(p));
}

uc* vectorize(){
	uc *img = new uc[SIZE*SIZE];
	for (size_t i = 0; i < SIZE; i++)
		for (size_t j = 0; j < SIZE; j++){
			size_t pos = (i*SIZE) + j;
			img[pos] = image1[i][j];
			//minimo = MIN_(minimo,img[pos]);
			//maximo = MAX_(maximo,img[pos]);
		}
	return img;
}

void vector_to_matrix( uc* img){
	for (size_t i = 0; i < SIZE; i++)
		for (size_t j = 0; j < SIZE; j++){
			size_t pos = SIZE*i + j;
			image2[i][j] = img[pos];
		}
}

//unsigned char image1[MAX_IMAGESIZE][MAX_IMAGESIZE]

int main(void)
{
	load_image_data();  
	uc *dev_img1;
	uc *dev_img2;

	uc *img1 = vectorize();	
	uc *img2 = new uc[SIZE*SIZE];
	
	hipMalloc((void**)&dev_img1, SIZE*SIZE*sizeof(uc));
	hipMalloc((void**)&dev_img2, SIZE*SIZE*sizeof(uc));

	hipMemcpy( dev_img1, img1, SIZE*SIZE*sizeof(uc), hipMemcpyHostToDevice);
	

	clock_t kerneltime=clock(); 
	// Haciendo el calculo de tiempo en el device
	kernel <<<N,L>>>(dev_img1, dev_img2,0.0,255.0);
	hipDeviceSynchronize();
	printf("\nsvm kernel time is:%f\n",((double)clock()-kerneltime)/CLOCKS_PER_SEC); 

	hipMemcpy(img2, dev_img2, SIZE*SIZE*sizeof(uc), hipMemcpyDeviceToHost);

	vector_to_matrix(img2);

	x_size2 = SIZE;
	y_size2 = SIZE;
	
	save_image_data();

	return 0;
}
